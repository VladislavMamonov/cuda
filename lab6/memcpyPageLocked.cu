#include "hip/hip_runtime.h"
#include <iostream>


#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)


using namespace std;



#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }}



__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}



int main()
{
    hipDeviceProp_t prop;
    int whichDevice;

    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        cout << "Device does not support overlapping" << endl;
        return 0;
    }

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    hipHostAlloc((void**)&dev_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&dev_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&dev_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    hipStream_t stream;
    hipStreamCreate(&stream);

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        kernel <<< N / 256, 256, 0, stream >>> (dev_a, dev_b, dev_c);
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "time: " << elapsedTime << " ms" << endl;
}
