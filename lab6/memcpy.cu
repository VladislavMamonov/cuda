#include <iostream>


#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 10)


using namespace std;



void memcpy()
{
    int *dev;
    int *host = new int[FULL_DATA_SIZE];

    hipMalloc((void**)&dev, FULL_DATA_SIZE * sizeof(int));

    for (int i = 0; i < FULL_DATA_SIZE; i++)
        host[i] = rand();

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMemcpy(dev, host, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "host->device elapsed time: " << elapsedTime << " ms" << endl;

    hipEventRecord(start, 0);
    hipMemcpy(host, dev, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "device->host elapsed time: " << elapsedTime << " ms" << endl;
}



void memcpy_PageLocked()
{
    int *dev;
    int *host;

    hipMalloc((void**)&dev, FULL_DATA_SIZE * sizeof(int));
    hipHostAlloc((void**)&host, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++)
        host[i] = rand();

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMemcpy(dev, host, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "page-locked host->device elapsed time: " << elapsedTime << " ms" << endl;

    hipEventRecord(start, 0);
    hipMemcpy(host, dev, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "page-locked device->host elapsed time: " << elapsedTime << " ms" << endl;
}



int main()
{
    hipDeviceProp_t prop;
    int whichDevice;

    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        cout << "Device does not support overlapping" << endl;
        return 1;
    }

    memcpy();
    cout << endl;
    memcpy_PageLocked();

    return 0;
}
