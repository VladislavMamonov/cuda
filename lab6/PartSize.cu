#include "hip/hip_runtime.h"
#include <iostream>


using namespace std;



#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }}



__global__ void VecSum(float *A, float *B, float *C, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size)
        C[i] = A[i] + B[i];
}



__global__ void VecMul(float *A, float *B, float *C, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size)
        C[i] = A[i] * B[i];
}



int main(int argc, char* argv[])
{
    if (argc != 3) {
        cout << "launch parametrs: [vector size] [partSize]" << endl;
        return 1;
    }

    hipDeviceProp_t prop;
    int whichDevice;

    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        cout << "Device does not support overlapping" << endl;
        return 1;
    }

    int full_data_size = atoi(argv[1]);
    int partSize = atoi(argv[2]);

    if (full_data_size % partSize != 0) {
        cout << "The size of the data chunk must be a multiple of the full data size" << endl;
        return 1;
    }

    float *A = new float[full_data_size];
    float *B = new float[full_data_size];
    float *C = new float[full_data_size];

    float *dev_a, *dev_b, *dev_c;
    hipHostAlloc((void**)&dev_a, full_data_size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&dev_b, full_data_size * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&dev_c, full_data_size * sizeof(int), hipHostMallocDefault);

    hipStream_t stream;
    hipStreamCreate(&stream);

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for (int i = 0; i < full_data_size; i += partSize) {
        hipMemcpyAsync(dev_a, A + i, partSize * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, B + i, partSize * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_c, C + i, partSize * sizeof(int), hipMemcpyHostToDevice, stream);
        VecMul <<< partSize / 256, 256, 0, stream >>> (dev_a, dev_b, dev_c, full_data_size);
        hipMemcpyAsync(C + i, dev_c, partSize * sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "time: " << elapsedTime << " ms" << endl;

    return 0;
}
