#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

#define SH_DIM 32


#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }}


__global__ void transpose(float *A, float *B)
{
    __shared__ float buffer_s[SH_DIM][SH_DIM];

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int N = blockDim.x * gridDim.x;

    buffer_s[threadIdx.y][threadIdx.x] = A[i + j * N];
    __syncthreads();

    i = threadIdx.x + blockIdx.y * blockDim.x;
    j = threadIdx.y + blockIdx.x * blockDim.y;
    B[i + j * N] = buffer_s[threadIdx.x][threadIdx.y];
}


void InitMatrix(float *A, float *B, int size)
{
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++) {
            int k = size * i + j;
            A[k] = k;
            B[k] = 0;
        }
}


void printMatrix(float *C, int size)
{
    for (int i = 0; i < size * size; i++)
        cout << C[i] << "\t";
    cout << endl;
}


double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);

    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}


int main(int argc, char* argv[])
{
    if (argc != 4) {
	    cout << "launch parametrs: [matrix size] [threads_x] [threads_y]" << endl;
        return 1;
    }

    int size = atoi(argv[1]);
    int threads_per_block_x = atoi(argv[2]);
    int threads_per_block_y = atoi(argv[3]);

    srand(time(NULL));

    float *A = new float[size * size];
    float *B = new float[size * size];

    float *dev_A, *dev_B;

    hipMalloc((void**)&dev_A, size * size * sizeof(float));
    hipMalloc((void**)&dev_B, size * size * sizeof(float));

    InitMatrix(A, B, size);

    dim3 threads(threads_per_block_x, threads_per_block_y);
    dim3 blocks(size / threads.x, size / threads.y);

    hipMemcpy(dev_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    transpose <<< blocks, threads >>> (dev_A, dev_B);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(B, dev_B, size * size * sizeof(float), hipMemcpyDeviceToHost);

    //printMatrix(B, size);
    cout << "time: " << elapsedTime << " ms" << endl;

    delete [] A; delete [] B;
    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(dev_A); hipFree(dev_B);

    return 0;
}
