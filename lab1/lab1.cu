#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

#define SIZE 8000000
#define THREADS_PER_BLOCK 64


__global__ void VecSum(float *A, float *B, float *C)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    C[i] = A[i] + B[i];
}


void printVec(float *C)
{
    for (int i = 0; i < SIZE; i++)
        cout << C[i] << "\t";
    cout << endl;
}


double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);

    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}


int main()
{
    srand(time(NULL));

    float *A = new float[SIZE];
    float *B = new float[SIZE];
    float *C = new float[SIZE];

    float *dev_A, *dev_B, *dev_C;

    hipMalloc((void**)&dev_A, SIZE * sizeof(float));
    hipMalloc((void**)&dev_B, SIZE * sizeof(float));
    hipMalloc((void**)&dev_C, SIZE * sizeof(float));

    for (int i = 0; i < SIZE; i++) {
        A[i] = rand();
        B[i] = rand();
    }

    hipMemcpy(dev_A, A, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_C, C, SIZE * sizeof(float), hipMemcpyHostToDevice);

    int blockTotal = SIZE / THREADS_PER_BLOCK;

    double time = wtime();
    VecSum <<< blockTotal, THREADS_PER_BLOCK >>> (dev_A, dev_B, dev_C);
    hipDeviceSynchronize();
    time = wtime() - time;

    hipMemcpy(C, dev_C, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    cout << "time: " << time << endl;
    //printVec(C);

    delete [] A; delete [] B; delete [] C;
    hipFree(dev_A); hipFree(dev_B); hipFree(dev_C);

    return 0;
}
