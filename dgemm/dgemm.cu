#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;


int threads_per_block = 8;


#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }}


__global__ void dgemm(float *A, float *B, float *C, int threads_per_block, int n)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    int ia = n * (threads_per_block * by + ty);    // Номер строки из A
    int ib = threads_per_block * bx + tx;          // Номер столбца из B
    int ic = ia + ib;                       // Номер элемента из C

    for (int k = 0; k < n; k++)
        sum += A[ia + k] * B[ib + k * n];
    C[ic] = sum;
}


void InitMatrix(float *A, float *B, float *C, int size)
{
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++) {
            int k = size * i + j;
            A[k] = rand();
            B[k] = rand();
            C[k] = 0.0;
        }
}


void printMatrix(float *C, int size)
{
    for (int i = 0; i < size; i++)
        cout << C[i] << "\t";
    cout << endl;
}


double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);

    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}


int main(int argc, char* argv[])
{
    if (argc != 4) {
	    cout << "launch parametrs: [matrix size] [threads_x] [threads_y]" << endl;
        return 1;
    }

    int size = atoi(argv[1]);
    int threads_per_block_x = atoi(argv[2]);
    int threads_per_block_y = atoi(argv[3]);

    srand(time(NULL));

    float *A = new float[size * size];
    float *B = new float[size * size];
    float *C = new float[size * size];

    float *dev_A, *dev_B, *dev_C;

    hipMalloc((void**)&dev_A, size * size * sizeof(float));
    hipMalloc((void**)&dev_B, size * size * sizeof(float));
    hipMalloc((void**)&dev_C, size * size * sizeof(float));

    InitMatrix(A, B, C, size);

    dim3 threads(threads_per_block_x, threads_per_block_y);
    dim3 blocks(size / threads.x, size / threads.y);

    hipMemcpy(dev_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_C, C, size * size * sizeof(float), hipMemcpyHostToDevice);

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    dgemm <<< blocks, threads >>> (dev_A, dev_B, dev_C, threads_per_block, size);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(C, dev_C, size * size * sizeof(float), hipMemcpyDeviceToHost);

    cout << "time: " << elapsedTime << " ms" << endl;
    //printMatrix(C, size);

    delete [] A; delete [] B; delete [] C;
    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(dev_A); hipFree(dev_B); hipFree(dev_C);

    return 0;
}
