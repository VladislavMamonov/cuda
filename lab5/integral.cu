#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M_PI 3.14159265358979323846
#define COEF 48
#define VERTCOUNT COEF*COEF*2
#define RADIUS 10.0f
#define FGSIZE 20
#define FGSHIFT FGSIZE / 2
#define IMIN(A, B) (A < B ? A : B)
#define THREADSPERBLOCK 256
#define BLOCKSPERGRID IMIN(32, (VERTCOUNT + THREADSPERBLOCK - 1) / THREADSPERBLOCK)


typedef float(*ptr_f)(float, float, float);

struct Vertex {
    float x, y, z;
};

__constant__ Vertex vert[VERTCOUNT];


float func(float x, float y, float z)
{
    return (0.5 * sqrtf(15.0 / M_PI)) * (0.5 * sqrtf(15.0 / M_PI))
        * z * z * y * y * sqrtf(1.0f - z * z / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS / RADIUS;
}



float check(Vertex *v, ptr_f f)
{
    float sum = 0.0f;

    for (int i = 0; i < VERTCOUNT; ++i)
        sum += f(v[i].x, v[i].y, v[i].z);

    return sum;
}



void calc_f(float *arr_f, int x_size, int y_size, int z_size, ptr_f f)
{
    for (int x = 0; x < x_size; ++x)
        for (int y = 0; y < y_size; ++y)
            for (int z = 0; z < z_size; ++z)
                arr_f[z_size * (x * y_size + y) + z] = f(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
}



void init_vertices()
{
    Vertex *temp_vert = (Vertex *)malloc(sizeof(Vertex) * VERTCOUNT);
    int i = 0;
    for (int iphi = 0; iphi < 2 * COEF; ++iphi) {
        for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i) {
            float phi = iphi * M_PI / COEF;
            float psi = ipsi * M_PI / COEF;
            temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
            temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
            temp_vert[i].z = RADIUS * cosf(psi);
        }
    }

    printf("sumcheck = %f\n", check(temp_vert, &func) * M_PI * M_PI / COEF / COEF);
    hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) * VERTCOUNT, 0, hipMemcpyHostToDevice);

    free(temp_vert);
}


__device__ float Trilinear_Interpolation(float x, float y, float z, float *arr) {   //Трилинейная интерполяция
    float res = 0.0f;
    float x_r[2], y_r[2], z_r[2];
    x_r[0] = x - 1;
    x_r[1] = x + 1;
    y_r[0] = y - 1;
    y_r[1] = y + 1;
    z_r[0] = z - 1;
    z_r[1] = z + 1;
    float denominator = (x_r[1] - x_r[0]) * (y_r[1] - y_r[0]) * (z_r[1] - z_r[0]);
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[0]) + (int)z_r[0]] * (x_r[1] - x) * (y_r[1] - y) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[0]) + (int)z_r[1]] * (x_r[1] - x) * (y_r[1] - y) * (z - z_r[0])) / denominator;
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[1]) + (int)z_r[0]] * (x_r[1] - x) * (y - y_r[0]) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[0] * FGSIZE + (int)y_r[1]) + (int)z_r[1]] * (x_r[1] - x) * (y - y_r[0]) * (z - z_r[0])) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[0]) + (int)z_r[0]] * (x - x_r[0]) * (y_r[1] - y) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[0]) + (int)z_r[1]] * (x - x_r[0]) * (y_r[1] - y) * (z - z_r[0])) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[1]) + (int)z_r[0]] * (x - x_r[0]) * (y - y_r[0]) * (z_r[1] - z)) / denominator;
    res += (arr[FGSIZE * ((int)x_r[1] * FGSIZE + (int)y_r[1]) + (int)z_r[1]] * (x - x_r[0]) * (y - y_r[0]) * (z - z_r[0])) / denominator;
    //printf("%f\n", res);
    return res;
}


__global__ void kernel(float *a, float *val)
{
    __shared__ float cache[THREADSPERBLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float x = vert[tid].x + FGSHIFT + 0.5f;
    float y = vert[tid].y + FGSHIFT + 0.5f;
    float z = vert[tid].z + FGSHIFT + 0.5f;
    cache[cacheIndex] = Trilinear_Interpolation(x, y, z, val);

    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (cacheIndex < s)
            cache[cacheIndex] += cache[cacheIndex + s];
        __syncthreads();
    }

    if (cacheIndex == 0)
        a[blockIdx.x] = cache[0];
}



int main()
{
    float *arr = (float *)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    float *sum = (float*)malloc(sizeof(float) * BLOCKSPERGRID);
    float *sum_dev;
    float *values;
    init_vertices();
    calc_f(arr, FGSIZE, FGSIZE, FGSIZE, &func);

    hipMalloc((void**)&sum_dev, sizeof(float) * BLOCKSPERGRID);
    hipMalloc((void**)&values, sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    hipMemcpy(values, arr, sizeof(float) * FGSIZE * FGSIZE * FGSIZE, hipMemcpyHostToDevice);

    kernel <<< BLOCKSPERGRID,THREADSPERBLOCK >>> (sum_dev, values);
    hipDeviceSynchronize();
    hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost);

    float s = 0.0f;
    for (int i = 0; i < BLOCKSPERGRID; ++i)
        s += sum[i];
    printf("sum = %f\n", s * M_PI * M_PI / COEF / COEF);

    hipFree(sum_dev);
    free(sum);
    free(arr);

    return 0;
}
