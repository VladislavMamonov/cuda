#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;


#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }}


__global__ void VecSum(float *A, float *B, float *C, int size)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size)
        C[i] = A[i] + B[i];
}


void printVec(float *C, int size)
{
    for (int i = 0; i < size; i++)
        cout << C[i] << "\t";
    cout << endl;
}


int main(int argc, char* argv[])
{
    if (argc != 3) {
        cout << "launch parametrs: [vector size] [threads per block]" << endl;
        return 1;
    }

    int size = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);

    srand(time(NULL));

    float *A = new float[size];
    float *B = new float[size];
    float *C = new float[size];

    float *dev_A, *dev_B, *dev_C;

    hipMalloc((void**)&dev_A, size * sizeof(float));
    hipMalloc((void**)&dev_B, size * sizeof(float));
    hipMalloc((void**)&dev_C, size * sizeof(float));

    for (int i = 0; i < size; i++) {
        A[i] = rand();
        B[i] = rand();
    }

    hipMemcpy(dev_A, A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_C, C, size * sizeof(float), hipMemcpyHostToDevice);

    int blockTotal = ceilf(float(size) / float(threads_per_block));
    cout << "Block total: " << blockTotal << endl;
    cout << "Threads per block : " << threads_per_block << endl;
    cout << "Threads total: " << blockTotal * threads_per_block << endl;

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    VecSum <<< blockTotal, threads_per_block >>> (dev_A, dev_B, dev_C, size);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(C, dev_C, size * sizeof(float), hipMemcpyDeviceToHost);

    cout << "time: " << elapsedTime << " ms" << endl;
    //printVec(C, size);

    delete [] A; delete [] B; delete [] C;
    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(dev_A); hipFree(dev_B); hipFree(dev_C);

    return 0;
}
