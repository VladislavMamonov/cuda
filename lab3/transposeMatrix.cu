#include <iostream>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;


#define CUDA_CHECK_RETURN(value) {\
    hipError_t _m_cudaStat = value;\
    if (_m_cudaStat != hipSuccess) {\
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
        exit(1);\
    }}


__global__ void transpose(float *A, float *B, int n)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = tx + bx * blockDim.x;
    int j = ty + by * blockDim.y;

    B[j * n + i] = A[i * n + j];
}


void InitMatrix(float *A, float *B, int size)
{
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++) {
            int k = size * i + j;
            A[k] = k;
            B[k] = 0;
        }
}


void printMatrix(float *C, int size)
{
    for (int i = 0; i < size * size; i++)
        cout << C[i] << "\t";
    cout << endl;
}


double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);

    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}


int main(int argc, char* argv[])
{
    if (argc != 4) {
	    cout << "launch parametrs: [matrix size] [threads_x] [threads_y]" << endl;
        return 1;
    }

    int size = atoi(argv[1]);
    int threads_per_block_x = atoi(argv[2]);
    int threads_per_block_y = atoi(argv[3]);

    srand(time(NULL));

    float *A = new float[size * size];
    float *B = new float[size * size];

    float *dev_A, *dev_B;

    hipMalloc((void**)&dev_A, size * size * sizeof(float));
    hipMalloc((void**)&dev_B, size * size * sizeof(float));

    InitMatrix(A, B, size);

    dim3 threads(threads_per_block_x, threads_per_block_y);
    dim3 blocks(size / threads.x, size / threads.y);

    hipMemcpy(dev_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);

    float elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    transpose <<< blocks, threads >>> (dev_A, dev_B, size);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipGetLastError());
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(B, dev_B, size * size * sizeof(float), hipMemcpyDeviceToHost);

    printMatrix(B, size);
    cout << "time: " << elapsedTime << " ms" << endl;

    delete [] A; delete [] B;
    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(dev_A); hipFree(dev_B);

    return 0;
}
